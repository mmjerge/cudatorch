#include "hip/hip_runtime.h"
/**
 * Matrix Multiplication Optimization with CUDA
 * 
 * This file contains multiple implementations of matrix multiplication:
 * 1. Naive implementation
 * 2. Shared memory optimization
 * 3. Tensor cores (WMMA) implementation
 * 4. cuBLAS implementation
 * 5. CUTLASS implementation
 *
 * Benchmarking across RTX 2080 Ti, A100, and H100 GPUs
 * Testing different matrix sizes and measuring throughput
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>
#include <chrono>
#include <vector>
#include <string>
#include <fstream>
#include <algorithm>
#include <hipblas.h>
#include <string.h>
#include <getopt.h>

// For CUTLASS implementation
#include <cutlass/gemm/device/gemm.h>

#define SMALL_SIZE 32
#define MEDIUM_SIZE 1024
#define LARGE_SIZE 8192
#define NON_SQUARE_M 1024
#define NON_SQUARE_N 2048
#define NON_SQUARE_K 1024

struct GPUInfo {
    char name[256];
    int major;
    int minor;
    bool hasTensorCores;
};

struct PerfResult {
    std::string gpuName;
    std::string implementation;
    int m;
    int n;
    int k;
    float executionTimeMs;
    float throughputGFlops;
};

// =============== KERNEL IMPLEMENTATIONS ===============

__global__ void matrixMulNaive(float *a, float *b, float *c, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        float sum = 0.0f;
        for (int i = 0; i < k; i++) {
            sum += a[row * k + i] * b[i * n + col];
        }
        c[row * n + col] = sum;
    }
}

__global__ void matrixMulShared(float *a, float *b, float *c, int m, int n, int k) {
    const int TILE_SIZE = 32;
    
    __shared__ float s_a[TILE_SIZE][TILE_SIZE];
    __shared__ float s_b[TILE_SIZE][TILE_SIZE];
    
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;
    
    float sum = 0.0f;
    
    for (int t = 0; t < (k + TILE_SIZE - 1) / TILE_SIZE; t++) {
        if (row < m && t * TILE_SIZE + tx < k) {
            s_a[ty][tx] = a[row * k + t * TILE_SIZE + tx];
        } else {
            s_a[ty][tx] = 0.0f;
        }
        
        if (t * TILE_SIZE + ty < k && col < n) {
            s_b[ty][tx] = b[(t * TILE_SIZE + ty) * n + col];
        } else {
            s_b[ty][tx] = 0.0f;
        }
        
        __syncthreads();
        
        for (int i = 0; i < TILE_SIZE; i++) {
            sum += s_a[ty][i] * s_b[i][tx];
        }
        
        __syncthreads();
    }
    
    if (row < m && col < n) {
        c[row * n + col] = sum;
    }
}

__global__ void convertToHalfKernel(float *in, half *out, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        out[idx] = __float2half(in[idx]);
    }
}

__global__ void matrixMulTensorCores(half *a, half *b, float *c, int m, int n, int k) {
    // WMMA dimensions
    const int WMMA_M = 16;
    const int WMMA_N = 16;
    const int WMMA_K = 16;
    
    using namespace nvcuda::wmma;
    
    fragment<matrix_a, WMMA_M, WMMA_N, WMMA_K, half, row_major> a_frag;
    fragment<matrix_b, WMMA_M, WMMA_N, WMMA_K, half, row_major> b_frag;
    fragment<accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;
    
    int warpM = (blockIdx.y * blockDim.y + threadIdx.y) / 32;
    int warpN = (blockIdx.x * blockDim.x + threadIdx.x) / 32;
    
    if (warpM * WMMA_M < m && warpN * WMMA_N < n) {
        fill_fragment(c_frag, 0.0f);
        
        for (int i = 0; i < k; i += WMMA_K) {
            if (i < k) {
                load_matrix_sync(a_frag, a + warpM * WMMA_M * k + i, k);
                load_matrix_sync(b_frag, b + i * n + warpN * WMMA_N, n);
                mma_sync(c_frag, a_frag, b_frag, c_frag);
            }
        }
        
        store_matrix_sync(c + warpM * WMMA_M * n + warpN * WMMA_N, c_frag, n, nvcuda::wmma::mem_row_major);
    }
}

// =============== CUTLASS IMPLEMENTATION ===============

template <typename Gemm>
void runCutlassGemm(float *d_a, float *d_b, float *d_c, int m, int n, int k) {
    Gemm gemm_operator;

    typename Gemm::Arguments args(
        {m, n, k},                   // Problem dimensions (M, N, K)
        {d_a, k},                    // Tensor A (device pointer and leading dimension)
        {d_b, n},                    // Tensor B (device pointer and leading dimension)
        {d_c, n},                    // Tensor C (device pointer and leading dimension)
        {d_c, n},                    // Tensor D (device pointer and leading dimension)
        {1.0f, 0.0f}                 // alpha and beta
    );

    hipDeviceSynchronize();
    cutlass::Status status = gemm_operator(args);
    hipDeviceSynchronize();

    if (status != cutlass::Status::kSuccess) {
        printf("CUTLASS GEMM kernel failed: %s\n", cutlass::cutlassGetStatusString(status));
    }
}

void matrixMulCutlass(float *d_a, float *d_b, float *d_c, int m, int n, int k) {
    using ElementInputA = float;
    using ElementInputB = float;
    using ElementOutput = float;
    using ElementAccumulator = float;
    using ElementCompute = float;

    using LayoutInputA = cutlass::layout::RowMajor;
    using LayoutInputB = cutlass::layout::RowMajor;
    using LayoutOutput = cutlass::layout::RowMajor;

    int device;
    hipGetDevice(&device);
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);

    if (props.major >= 8) {
        printf("Using CUTLASS configuration for Ampere+ architecture\n");
        
        using CutlassGemm = cutlass::gemm::device::Gemm<
            ElementInputA, LayoutInputA,
            ElementInputB, LayoutInputB,
            ElementOutput, LayoutOutput,
            ElementAccumulator,
            cutlass::arch::OpClassSimt,     // Using SIMT architecture
            cutlass::arch::Sm80             // Target SM architecture
        >;
        
        runCutlassGemm<CutlassGemm>(d_a, d_b, d_c, m, n, k);
    }
    else if (props.major >= 7) {
        // Volta/Turing (SM70-75)
        printf("Using CUTLASS configuration for Volta/Turing architecture\n");
        
        using CutlassGemm = cutlass::gemm::device::Gemm<
            ElementInputA, LayoutInputA,
            ElementInputB, LayoutInputB,
            ElementOutput, LayoutOutput,
            ElementAccumulator,
            cutlass::arch::OpClassSimt,     // Using SIMT architecture
            cutlass::arch::Sm70             // Target SM architecture
        >;
        
        runCutlassGemm<CutlassGemm>(d_a, d_b, d_c, m, n, k);
    }
    else {
        // Pascal or older (SM60 or below)
        printf("Using CUTLASS configuration for Pascal or older architecture\n");
        
        using CutlassGemm = cutlass::gemm::device::Gemm<
            ElementInputA, LayoutInputA,
            ElementInputB, LayoutInputB,
            ElementOutput, LayoutOutput,
            ElementAccumulator,
            cutlass::arch::OpClassSimt,     // Using SIMT architecture
            cutlass::arch::Sm60             // Target SM architecture
        >;
        
        runCutlassGemm<CutlassGemm>(d_a, d_b, d_c, m, n, k);
    }
}

// CUTLASS Tensor Core implementation
void matrixMulCutlassTensorCores(float *d_a, float *d_b, float *d_c, int m, int n, int k) {
    // Check if we have a GPU that supports tensor cores
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);
    
    if (props.major >= 7) {
        printf("Using CUTLASS Tensor Core configuration\n");
        
        // Allocate and convert to half precision
        half *d_a_half, *d_b_half;
        hipMalloc(&d_a_half, m * k * sizeof(half));
        hipMalloc(&d_b_half, k * n * sizeof(half));
        
        // Convert float to half
        dim3 block(256);
        dim3 grid_a((m * k + block.x - 1) / block.x);
        dim3 grid_b((k * n + block.x - 1) / block.x);
        
        convertToHalfKernel<<<grid_a, block>>>(d_a, d_a_half, m * k);
        convertToHalfKernel<<<grid_b, block>>>(d_b, d_b_half, k * n);
        
        // Determine architecture-specific configuration
        if (props.major >= 8) {
            // Ampere or newer
            using ElementInputA = cutlass::half_t;
            using ElementInputB = cutlass::half_t;
            using ElementOutput = float;
            using ElementAccumulator = float;
            
            using LayoutInputA = cutlass::layout::RowMajor;
            using LayoutInputB = cutlass::layout::RowMajor;
            using LayoutOutput = cutlass::layout::RowMajor;
            
            using CutlassGemmTensorOp = cutlass::gemm::device::Gemm<
                ElementInputA, LayoutInputA,
                ElementInputB, LayoutInputB,
                ElementOutput, LayoutOutput,
                ElementAccumulator,
                cutlass::arch::OpClassTensorOp,
                cutlass::arch::Sm80
            >;
            
            dim3 block_tc(128, 4);
            dim3 grid_tc((n + 16 - 1) / 16, (m + 16 - 1) / 16);
            matrixMulTensorCores<<<grid_tc, block_tc>>>(d_a_half, d_b_half, d_c, m, n, k);
        }
        else {
            // Volta/Turing
            dim3 block_tc(128, 4);
            dim3 grid_tc((n + 16 - 1) / 16, (m + 16 - 1) / 16);
            matrixMulTensorCores<<<grid_tc, block_tc>>>(d_a_half, d_b_half, d_c, m, n, k);
        }
        
        // Clean up
        hipFree(d_a_half);
        hipFree(d_b_half);
    }
    else {
        printf("This GPU does not support tensor cores. Running standard CUTLASS GEMM.\n");
        matrixMulCutlass(d_a, d_b, d_c, m, n, k);
    }
}

// =============== HELPER FUNCTIONS ===============

void checkCudaError(hipError_t error, const char *message) {
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error: %s: %s\n", message, hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

void checkCublasError(hipblasStatus_t status, const char *message) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS error: %s: %d\n", message, status);
        exit(EXIT_FAILURE);
    }
}

void initializeMatrix(float *matrix, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        matrix[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

void convertToHalf(float *src, half *dst, int size) {
    for (int i = 0; i < size; i++) {
        dst[i] = __float2half(src[i]);
    }
}

bool verifyResults(float *a, float *b, float *c, int m, int n, int k) {
    float *verification = (float*)malloc(m * n * sizeof(float));
    
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            verification[i * n + j] = 0.0f;
            for (int p = 0; p < k; p++) {
                verification[i * n + j] += a[i * k + p] * b[p * n + j];
            }
        }
    }
    
    const float epsilon = 1e-2;  
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            if (fabs(verification[i * n + j] - c[i * n + j]) > epsilon) {
                printf("Verification failed at [%d, %d]: Expected %f, got %f\n", 
                       i, j, verification[i * n + j], c[i * n + j]);
                free(verification);
                return false;
            }
        }
    }
    
    free(verification);
    return true;
}

// Calculate GFLOPs for matrix multiplication
float calculateGFlops(int m, int n, int k, float timeMs) {
    // Each matrix multiply-add is 2 operations
    // Total operations = m * n * k * 2
    float operations = 2.0f * static_cast<float>(m) * static_cast<float>(n) * static_cast<float>(k);
    float timeS = timeMs / 1000.0f;
    return operations / (timeS * 1e9);
}

// Detect available GPUs and return their info
std::vector<GPUInfo> detectGPUs() {
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    
    std::vector<GPUInfo> gpus;
    
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);
        
        GPUInfo gpu;
        strcpy(gpu.name, deviceProp.name);
        gpu.major = deviceProp.major;
        gpu.minor = deviceProp.minor;
        gpu.hasTensorCores = (deviceProp.major >= 7);
        
        gpus.push_back(gpu);
    }
    
    return gpus;
}

bool isTargetGPU(const char* name) {
    // Check if the GPU is one of our target GPUs
    return (strstr(name, "RTX 2080 Ti") != NULL ||
            strstr(name, "A100") != NULL ||
            strstr(name, "H100") != NULL);
}

// Function to parse command line arguments
void parseArgs(int argc, char **argv, std::string &targetGPU) {
    const struct option long_options[] = {
        {"gpu", required_argument, 0, 'g'},
        {"help", no_argument, 0, 'h'},
        {0, 0, 0, 0}
    };

    int opt;
    int option_index = 0;
    
    while ((opt = getopt_long(argc, argv, "g:h", long_options, &option_index)) != -1) {
        switch (opt) {
            case 'g':
                targetGPU = optarg;
                break;
            case 'h':
                printf("Usage: %s [OPTIONS]\n", argv[0]);
                printf("Options:\n");
                printf("  -g, --gpu=GPU_NAME    Specify GPU to use (e.g., \"RTX 2080 Ti\", \"A100\", \"H100\")\n");
                printf("  -h, --help            Display this help message\n");
                exit(0);
            default:
                fprintf(stderr, "Try '%s --help' for more information.\n", argv[0]);
                exit(1);
        }
    }
}

// =============== CUBLAS IMPLEMENTATION ===============

void matrixMulCublas(hipblasHandle_t handle, float *d_a, float *d_b, float *d_c, int m, int n, int k) {
    const float alpha = 1.0f;
    const float beta = 0.0f;
    
    // Note: cuBLAS uses column-major ordering, so we compute B*A instead of A*B
    checkCublasError(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                                  n, m, k, 
                                  &alpha, 
                                  d_b, n, 
                                  d_a, k, 
                                  &beta, 
                                  d_c, n), 
                     "Executing cuBLAS SGEMM");
}

// =============== TESTING FRAMEWORK ===============

struct NaiveParams {
    float *d_a, *d_b, *d_c;
};

struct SharedParams {
    float *d_a, *d_b, *d_c;
};

struct TensorParams {
    half *d_a, *d_b;
    float *d_c;
};

struct CublasParams {
    hipblasHandle_t handle;
    float *d_a, *d_b, *d_c;
};

struct CutlassParams {
    float *d_a, *d_b, *d_c;
};

void naiveBenchmark(void* p, int m, int n, int k) {
    NaiveParams* params = (NaiveParams*)p;
    dim3 blockDim(32, 32);
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x, (m + blockDim.y - 1) / blockDim.y);
    matrixMulNaive<<<gridDim, blockDim>>>(params->d_a, params->d_b, params->d_c, m, n, k);
    checkCudaError(hipGetLastError(), "Launching naive kernel");
    hipDeviceSynchronize();
}

void sharedBenchmark(void* p, int m, int n, int k) {
    SharedParams* params = (SharedParams*)p;
    dim3 blockDim(32, 32);
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x, (m + blockDim.y - 1) / blockDim.y);
    matrixMulShared<<<gridDim, blockDim>>>(params->d_a, params->d_b, params->d_c, m, n, k);
    checkCudaError(hipGetLastError(), "Launching shared memory kernel");
    hipDeviceSynchronize();
}

void tensorBenchmark(void* p, int m, int n, int k) {
    TensorParams* params = (TensorParams*)p;
    dim3 blockDim(128, 4);
    dim3 gridDim((n + 16 - 1) / 16, (m + 16 - 1) / 16);
    matrixMulTensorCores<<<gridDim, blockDim>>>(params->d_a, params->d_b, params->d_c, m, n, k);
    checkCudaError(hipGetLastError(), "Launching tensor cores kernel");
    hipDeviceSynchronize();
}

void cublasBenchmark(void* p, int m, int n, int k) {
    CublasParams* params = (CublasParams*)p;
    matrixMulCublas(params->handle, params->d_a, params->d_b, params->d_c, m, n, k);
    hipDeviceSynchronize();
}

void cutlassBenchmark(void* p, int m, int n, int k) {
    CutlassParams* params = (CutlassParams*)p;
    matrixMulCutlass(params->d_a, params->d_b, params->d_c, m, n, k);
    hipDeviceSynchronize();
}

void cutlassTensorBenchmark(void* p, int m, int n, int k) {
    CutlassParams* params = (CutlassParams*)p;
    matrixMulCutlassTensorCores(params->d_a, params->d_b, params->d_c, m, n, k);
    hipDeviceSynchronize();
}

PerfResult runBenchmark(const char* gpuName, const char* implName, 
                      void (*benchmark)(void*, int, int, int), 
                      void* params, int m, int n, int k, bool verify) {
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsed_time;
    
    hipEventRecord(start);
    benchmark(params, m, n, k);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    
    float gflops = calculateGFlops(m, n, k, elapsed_time);
    
    PerfResult result;
    result.gpuName = gpuName;
    result.implementation = implName;
    result.m = m;
    result.n = n;
    result.k = k;
    result.executionTimeMs = elapsed_time;
    result.throughputGFlops = gflops;
    
    printf("%-15s %-25s %5d x %5d x %5d: %10.2f ms (%10.2f GFlops)\n", 
           gpuName, implName, m, n, k, elapsed_time, gflops);
    
    return result;
}

void saveResultsToCSV(const std::vector<PerfResult>& results, const char* filename) {
    std::ofstream file(filename);
    if (!file.is_open()) {
        fprintf(stderr, "Error: Could not open file %s for writing\n", filename);
        return;
    }
    
    file << "GPU,Implementation,M,N,K,Time_ms,Throughput_GFlops\n";
    
    for (const auto& result : results) {
        file << result.gpuName << ","
             << result.implementation << ","
             << result.m << ","
             << result.n << ","
             << result.k << ","
             << result.executionTimeMs << ","
             << result.throughputGFlops << "\n";
    }
    
    file.close();
}

// =============== MAIN FUNCTION ===============

int main(int argc, char** argv) {
    std::string targetGPU = "";  // Empty string means all GPUs
    parseArgs(argc, argv, targetGPU);
    
    std::vector<GPUInfo> gpus = detectGPUs();
    std::vector<PerfResult> allResults;
    
    if (gpus.empty()) {
        fprintf(stderr, "No CUDA-capable devices found\n");
        return EXIT_FAILURE;
    }
    
    printf("Detected GPUs:\n");
    for (const auto& gpu : gpus) {
        printf("  %s (SM %d.%d)\n", gpu.name, gpu.major, gpu.minor);
    }
    
    if (!targetGPU.empty()) {
        printf("\nFiltering to run only on: %s\n", targetGPU.c_str());
    }
    
    struct MatrixSize {
        int m, n, k;
        const char* name;
    };
    
    MatrixSize sizes[] = {
        {SMALL_SIZE, SMALL_SIZE, SMALL_SIZE, "Small (32x32)"},
        {MEDIUM_SIZE, MEDIUM_SIZE, MEDIUM_SIZE, "Medium (1024x1024)"},
        {LARGE_SIZE, LARGE_SIZE, LARGE_SIZE, "Large (8192x8192)"},
        {NON_SQUARE_M, NON_SQUARE_N, NON_SQUARE_K, "Non-square (1024x2048)"}
    };
    
    // Loop through each GPU
    for (const auto& gpu : gpus) {
        if (!isTargetGPU(gpu.name) || (!targetGPU.empty() && strstr(gpu.name, targetGPU.c_str()) == NULL)) {
            printf("Skipping GPU: %s (not selected for testing)\n", gpu.name);
            continue;
        }
        
        printf("\n=== Testing on GPU: %s ===\n", gpu.name);
        
        int deviceId = -1;
        for (int i = 0; i < gpus.size(); i++) {
            if (strcmp(gpu.name, gpus[i].name) == 0) {
                deviceId = i;
                break;
            }
        }
        
        if (deviceId == -1) {
            fprintf(stderr, "Error: Could not find device ID for GPU %s\n", gpu.name);
            continue;
        }
        
        hipSetDevice(deviceId);
        
        hipblasHandle_t cublasHandle;
        checkCublasError(hipblasCreate(&cublasHandle), "Creating cuBLAS handle");
        
        for (const auto& size : sizes) {
            int m = size.m;
            int n = size.n;
            int k = size.k;
            
            printf("\n--- Testing %s matrices ---\n", size.name);
            
            bool isLargeMatrix = (m >= LARGE_SIZE || n >= LARGE_SIZE || k >= LARGE_SIZE);
            
            float *h_a = (float*)malloc(m * k * sizeof(float));
            float *h_b = (float*)malloc(k * n * sizeof(float));
            float *h_c = (float*)malloc(m * n * sizeof(float));
            half *h_a_half = (half*)malloc(m * k * sizeof(half));
            half *h_b_half = (half*)malloc(k * n * sizeof(half));
            
            srand(42);  
            initializeMatrix(h_a, m, k);
            initializeMatrix(h_b, k, n);
            
            convertToHalf(h_a, h_a_half, m * k);
            convertToHalf(h_b, h_b_half, k * n);
            
            float *d_a, *d_b, *d_c;
            half *d_a_half, *d_b_half;
            
            size_t bytes_a = m * k * sizeof(float);
            size_t bytes_b = k * n * sizeof(float);
            size_t bytes_c = m * n * sizeof(float);
            size_t bytes_a_half = m * k * sizeof(half);
            size_t bytes_b_half = k * n * sizeof(half);
            
            checkCudaError(hipMalloc(&d_a, bytes_a), "Allocating d_a");
            checkCudaError(hipMalloc(&d_b, bytes_b), "Allocating d_b");
            checkCudaError(hipMalloc(&d_c, bytes_c), "Allocating d_c");
            checkCudaError(hipMalloc(&d_a_half, bytes_a_half), "Allocating d_a_half");
            checkCudaError(hipMalloc(&d_b_half, bytes_b_half), "Allocating d_b_half");
            
            checkCudaError(hipMemcpy(d_a, h_a, bytes_a, hipMemcpyHostToDevice), "Copying h_a to d_a");
            checkCudaError(hipMemcpy(d_b, h_b, bytes_b, hipMemcpyHostToDevice), "Copying h_b to d_b");
            checkCudaError(hipMemcpy(d_a_half, h_a_half, bytes_a_half, hipMemcpyHostToDevice), "Copying h_a_half to d_a_half");
            checkCudaError(hipMemcpy(d_b_half, h_b_half, bytes_b_half, hipMemcpyHostToDevice), "Copying h_b_half to d_b_half");
            
            NaiveParams naiveParams = {d_a, d_b, d_c};
            SharedParams sharedParams = {d_a, d_b, d_c};
            TensorParams tensorParams = {d_a_half, d_b_half, d_c};
            CublasParams cublasParams = {cublasHandle, d_a, d_b, d_c};
            CutlassParams cutlassParams = {d_a, d_b, d_c};
            
            if (!isLargeMatrix) {
                PerfResult naiveResult = runBenchmark(gpu.name, "Naive", naiveBenchmark, &naiveParams, m, n, k, true);
                allResults.push_back(naiveResult);
                
                checkCudaError(hipMemcpy(h_c, d_c, bytes_c, hipMemcpyDeviceToHost), "Copying d_c to h_c (naive)");
                printf("Naive verification: %s\n", verifyResults(h_a, h_b, h_c, m, n, k) ? "PASSED" : "FAILED");
            }
            
            PerfResult sharedResult = runBenchmark(gpu.name, "Shared Memory", sharedBenchmark, &sharedParams, m, n, k, true);
            allResults.push_back(sharedResult);
            
            checkCudaError(hipMemcpy(h_c, d_c, bytes_c, hipMemcpyDeviceToHost), "Copying d_c to h_c (shared)");
            printf("Shared memory verification: %s\n", verifyResults(h_a, h_b, h_c, m, n, k) ? "PASSED" : "FAILED");
            
            if (gpu.hasTensorCores) {
                PerfResult tensorResult = runBenchmark(gpu.name, "Tensor Cores", tensorBenchmark, &tensorParams, m, n, k, true);
                allResults.push_back(tensorResult);
                
                checkCudaError(hipMemcpy(h_c, d_c, bytes_c, hipMemcpyDeviceToHost), "Copying d_c to h_c (tensor)");
                printf("Tensor cores verification: %s\n", verifyResults(h_a, h_b, h_c, m, n, k) ? "PASSED" : "FAILED");
            }
            
            PerfResult cublasResult = runBenchmark(gpu.name, "cuBLAS", cublasBenchmark, &cublasParams, m, n, k, true);
            allResults.push_back(cublasResult);
            
            checkCudaError(hipMemcpy(h_c, d_c, bytes_c, hipMemcpyDeviceToHost), "Copying d_c to h_c (cuBLAS)");
            printf("cuBLAS verification: %s\n", verifyResults(h_a, h_b, h_c, m, n, k) ? "PASSED" : "FAILED");
            
            PerfResult cutlassResult = runBenchmark(gpu.name, "CUTLASS", cutlassBenchmark, &cutlassParams, m, n, k, true);
            allResults.push_back(cutlassResult);
            
            checkCudaError(hipMemcpy(h_c, d_c, bytes_c, hipMemcpyDeviceToHost), "Copying d_c to h_c (CUTLASS)");
            printf("CUTLASS verification: %s\n", verifyResults(h_a, h_b, h_c, m, n, k) ? "PASSED" : "FAILED");
            
            if (gpu.hasTensorCores) {
                PerfResult cutlassTensorResult = runBenchmark(gpu.name, "CUTLASS Tensor Cores", cutlassTensorBenchmark, &cutlassParams, m, n, k, true);
                allResults.push_back(cutlassTensorResult);
                
                checkCudaError(hipMemcpy(h_c, d_c, bytes_c, hipMemcpyDeviceToHost), "Copying d_c to h_c (CUTLASS Tensor)");
                printf("CUTLASS Tensor verification: %s\n", verifyResults(h_a, h_b, h_c, m, n, k) ? "PASSED" : "FAILED");
            }
            
            hipFree(d_a);
            hipFree(d_b);
            hipFree(d_c);
            hipFree(d_a_half);
            hipFree(d_b_half);
            
            free(h_a);
            free(h_b);
            free(h_c);
            free(h_a_half);
            free(h_b_half);
        }
        
        hipblasDestroy(cublasHandle);
    }
    
    std::string csvFilename = "matrix_mul_performance";
    if (!targetGPU.empty()) {
        std::string gpuNameForFile = targetGPU;
        std::replace(gpuNameForFile.begin(), gpuNameForFile.end(), ' ', '_');
        csvFilename += "_" + gpuNameForFile;
    }
    csvFilename += ".csv";
    
    saveResultsToCSV(allResults, csvFilename.c_str());
    
    printf("\nResults saved to %s. Use this file to generate charts.\n", csvFilename.c_str());
    
    return 0;
}
